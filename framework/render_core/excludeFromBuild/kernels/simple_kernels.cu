#include "hip/hip_runtime.h"
﻿#pragma once

#include "simple_shared.h"

using namespace Shared;
using namespace SimpleShared;

RT_PIPELINE_LAUNCH_PARAMETERS PipelineLaunchParameters plp;

struct HitPointParameter
{
    float b1, b2;
    int32_t primIndex;

    CUDA_DEVICE_FUNCTION static HitPointParameter get()
    {
        HitPointParameter ret;
        float2 bc = optixGetTriangleBarycentrics();
        ret.b1 = bc.x;
        ret.b2 = bc.y;
        ret.primIndex = optixGetPrimitiveIndex();
        return ret;
    }
};

struct HitGroupSBTRecordData
{
    MaterialData matData;
    GeometryData geomData;

    CUDA_DEVICE_FUNCTION static const HitGroupSBTRecordData& get()
    {
        return *reinterpret_cast<HitGroupSBTRecordData*> (optixGetSbtDataPointer());
    }
};

#define PayloadSignature float3

CUDA_DEVICE_KERNEL void RT_RG_NAME (raygen)()
{
    uint2 launchIndex = make_uint2 (optixGetLaunchIndex().x, optixGetLaunchIndex().y);

    float x = static_cast<float> (launchIndex.x + 0.5f) / plp.imageSize.x;
    float y = static_cast<float> (plp.imageSize.y - launchIndex.y - 0.5f) / plp.imageSize.y;
    if (plp.pickingEnabled)
    {
        x = plp.pickingFragment.x / plp.imageSize.x;
        y = plp.pickingFragment.y / plp.imageSize.y;
    }
    plp.pickData->intanceID = ~0;
    plp.pickData->primitiveID = ~0;

    float vh = 2 * std::tan (plp.camera.fovY * 0.5f);
    float vw = plp.camera.aspect * vh;

    float3 origin = plp.camera.position;
    float3 direction = normalize (plp.camera.orientation * make_float3 (vw * (0.5f - x), vh * (y - 0.5f), 1));

    if (plp.pickingEnabled)
    {
        printf ("####xx Pick Data ####\n");
        printf ("Pick origin\n %f %f %f\n",
                origin.x, origin.y, origin.z);
        printf ("Pick direction\n %f %f %f\n",
                direction.x, direction.y, direction.z);
       
    }

    float3 color;
    optixu::trace<PayloadSignature> (
        plp.travHandle, origin, direction,
        0.0f, FLT_MAX, 0.0f, 0xFF, OPTIX_RAY_FLAG_NONE,
        RayType_Primary, NumRayTypes, RayType_Primary,
        color);

    plp.resultBuffer[launchIndex] = make_float4 (color, 1.0f);
}

CUDA_DEVICE_KERNEL void RT_MS_NAME (miss)()
{
    float3 color = make_float3 (0, 0, 0.1f);
    optixu::setPayloads<PayloadSignature> (&color);
}

CUDA_DEVICE_KERNEL void RT_CH_NAME (closesthit)()
{
    auto sbtr = HitGroupSBTRecordData::get();
    const GeometryData& geom = sbtr.geomData;
    const MaterialData& mat = sbtr.matData;

    auto hp = HitPointParameter::get();

    const Triangle& triangle = geom.triangleBuffer[hp.primIndex];
    const Vertex& v0 = geom.vertexBuffer[triangle.index0];
    const Vertex& v1 = geom.vertexBuffer[triangle.index1];
    const Vertex& v2 = geom.vertexBuffer[triangle.index2];

    if (plp.pickingEnabled)
    {
        PickingData* pickData = plp.pickData;
        pickData->primitiveID = hp.primIndex;
        pickData->intanceID = optixGetInstanceIndex();

        const float3 ray_orig = optixGetWorldRayOrigin();
        const float3 ray_dir = optixGetWorldRayDirection();
        const float ray_t = optixGetRayTmax();

        float3 spot = ray_orig + ray_t * ray_dir;
        pickData->hitPoint = spot;

        printf ("Hit point\n %f %f %f\n",
                spot.x, spot.y, spot.z);
        printf ("############################################\n");
        return;
    }

    float b0 = 1 - (hp.b1 + hp.b2);

    float3 sn = b0 * v0.normal + hp.b1 * v1.normal + hp.b2 * v2.normal;
    sn = normalize (optixTransformNormalFromObjectToWorldSpace (sn));

    float3 color = 0.5f * sn + make_float3 (0.5f);
    float3 albedo = mat.albedo;
    color.x *= albedo.x;
    color.y *= albedo.y;
    color.z *= albedo.z;

    optixu::setPayloads<PayloadSignature> (&color);
}
